#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void add(int *a, int *b, int *c, int n) {
    int i = threadIdx.x;
    if (i < n)
        c[i] = a[i] + b[i];
}

int main() {
    const int N = 5;
    int h_a[N] = {1, 2, 3, 4, 5};
    int h_b[N] = {10, 20, 30, 40, 50};
    int h_c[N];

    int *d_a, *d_b, *d_c;
    hipMalloc((void**)&d_a, N * sizeof(int));
    hipMalloc((void**)&d_b, N * sizeof(int));
    hipMalloc((void**)&d_c, N * sizeof(int));

    hipMemcpy(d_a, h_a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N * sizeof(int), hipMemcpyHostToDevice);

    add<<<1, N>>>(d_a, d_b, d_c, N);

    hipMemcpy(h_c, d_c, N * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < N; ++i)
        printf("%d + %d = %d\n", h_a[i], h_b[i], h_c[i]);

    hipFree(d_a); hipFree(d_b); hipFree(d_c);
    return 0;
}
