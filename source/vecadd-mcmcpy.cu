#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <time.h>

__global__ void Vec_add(
    const float x[], /* in */
    const float y[], /* in */
    float z[],       /* out */
    const int n      /* in */
)
{
    int my_elt = blockDim.x * blockIdx.x + threadIdx.x;

    /* total threads = blk_ct * th_per_blk pode ser > n */
    if (my_elt < n)
    {
        z[my_elt] = x[my_elt] + y[my_elt];
    }
} /* Vec_add */

void Serial_vec_add(
    const float x[], /* in */
    const float y[], /* in */
    float cz[],      /* out */
    const int n      /* in */
)
{
    for (int i = 0; i < n; i++)
    {
        cz[i] = x[i] + y[i];
    }
} /* Serial_vec_add */

void Get_args(
    const int argc,   /* in */
    char *argv[],     /* in */
    int *n_p,         /* out */
    int *blk_ct_p,    /* out */
    int *th_per_blk_p /* out */
)
{
    if (argc != 4)
    {
        fprintf(stderr, "Uso: %s <num_elementos> <num_blocos> <threads_por_bloco>\n", argv[0]);
        exit(EXIT_FAILURE);
    }

    *n_p = strtol(argv[1], NULL, 10);
    *blk_ct_p = strtol(argv[2], NULL, 10);
    *th_per_blk_p = strtol(argv[3], NULL, 10);

    if (*n_p > (*blk_ct_p) * (*th_per_blk_p))
    {
        fprintf(stderr, "Erro: num_elementos (%d) maior que total de threads (%d = %d * %d).\n",
                *n_p, (*blk_ct_p) * (*th_per_blk_p), *blk_ct_p, *th_per_blk_p);
        exit(EXIT_FAILURE);
    }
}

void Allocate_vectors(
    float** hx_p,     /* out */
    float** hy_p,     /* out */
    float** hz_p,     /* out */
    float** cz_p,     /* out */
    float** dx_p,     /* out */
    float** dy_p,     /* out */
    float** dz_p,     /* out */
    int n             /* in */
) 
{
    /* dx, dy, e dz são usados no dispositivo */
    hipMalloc(dx_p, n * sizeof(float));
    hipMalloc(dy_p, n * sizeof(float));
    hipMalloc(dz_p, n * sizeof(float));

    /* hx, hy, hz, cz são usados no host */
    *hx_p = (float*)malloc(n * sizeof(float));
    *hy_p = (float*)malloc(n * sizeof(float));
    *hz_p = (float*)malloc(n * sizeof(float));
    *cz_p = (float*)malloc(n * sizeof(float));

} /* Allocate_vectors */

double Two_norm_diff(
    const float z[],  /* in */
    const float cz[], /* in */
    const int n       /* in */
)
{
    double diff, sum = 0.0;

    for (int i = 0; i < n; i++)
    {
        diff = z[i] - cz[i];
        sum += diff * diff;
    }

    return sqrt(sum);

} /* Two_norm_diff */

void Free_vectors(
    float *x, /* in/out */
    float *y, /* in/out */
    float *z, /* in/out */
    float *cz /* in/out */
)
{
    /* Alocado com cudaMallocManaged */
    hipFree(x);
    hipFree(y);
    hipFree(z);

    /* Alocado com malloc */
    free(cz);

} /* Free_vectors */

void Init_vectors(float *x,
                  float *y,
                  const int n /* in */)
{
    int i;
    srand(time(NULL));
    for (i = 0; i < n; i++)
    {
        x[i] = (float)((double)rand() / RAND_MAX) * 1000;
        y[i] = (float)((double)rand() / RAND_MAX) * 1000;
    }
}

int main(int argc, char* argv[]) {
    int n, th_per_blk, blk_ct;
    float *hx, *hy, *hz, *cz; /* Vetores do Host */
    float *dx, *dy, *dz;     /* Vetores do Dispositivo */
    double diff_norm;

    Get_args(argc, argv, &n, &blk_ct, &th_per_blk);
    Allocate_vectors(&hx, &hy, &hz, &cz, &dx, &dy, &dz, n);
    Init_vectors(hx, hy, n);

    /* Copia os vetores x e y do host para o dispositivo */
    hipMemcpy(dx, hx, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dy, hy, n * sizeof(float), hipMemcpyHostToDevice);

    Vec_add<<<blk_ct, th_per_blk>>>(dx, dy, dz, n);

    /* Espera o kernel terminar e copia o resultado para o host */
    hipMemcpy(hz, dz, n * sizeof(float), hipMemcpyDeviceToHost);

    Serial_vec_add(hx, hy, cz, n);
    diff_norm = Two_norm_diff(hz, cz, n);
    printf("Norma-2 da diferença entre host e dispositivo = %e\n", diff_norm);

    Free_vectors(hx, hy, hz, cz);

    return 0;
} /* main */
